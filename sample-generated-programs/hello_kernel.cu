#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define imin(a,b) (a<b?a:b)

__global__ void vecAdd(int *x, int *y,int *z, int *scale){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	z[tid] = *scale *( x[tid] + y[tid]);
}

int main(){
	int a[5] = {1,2,3,4,5};
	int b[5] = {1,2,3,4,5};
	int c[5];

	int constant_scale = 4;
	int *dev_a;
	int *dev_b;
	int *dev_c;
	int *scale;

	/* Upon hitting map, this code should generate*/
	int threadsPerBlock = 512;
	int blocksPerGrid = ceil(5/512);

	hipInit(0);
	hipGetDeviceCount(&devCount);
	hipDeviceGet(&device, 0);

	int devMajor, devMinor;
  hipDeviceComputeCapability(&devMajor, &devMinor, device);
  std::cout << "Device Compute Capability: "<< devMajor << \".\" << devMinor << \"\n\";
  if (devMajor < 2) {
    std::cerr << \"ERROR: Device 0 is not SM 2.0 or greater\n\";
    return 1;
  }
	hipMalloc( (void**) &dev_a, 5*sizeof(int) );
   	hipMalloc( (void**) &dev_b, 5*sizeof(int) );
   	hipMalloc( (void**) &dev_c, 5*sizeof(int) );
   	hipMalloc( (void**) &scale, sizeof(int) );


   	hipMemcpy( dev_a, a, 5*sizeof(int), hipMemcpyHostToDevice ) ;
   	hipMemcpy( dev_b, b, 5*sizeof(int), hipMemcpyHostToDevice ) ;
   	hipMemcpy( scale, &constant_scale, sizeof(int), hipMemcpyHostToDevice );

   	vecAdd<<<blocksPerGrid,threadsPerBlock>>>( dev_a, dev_b, dev_c, scale);
 	hipMemcpy( c, dev_c, 5*sizeof(int),hipMemcpyDeviceToHost ) ;
 
 	printf("[");
 	for(int i = 0; i < 5; i ++){
 		if(i == 5 - 1){
 			printf("%d",c[i]);
 		}
 		else{
 			printf("%d,",c[i]);
 		}
 	}
 	printf("]");
	hipDeviceSynchronize();
	hipFree( dev_a ) ;
   	hipFree( dev_b ) ;
   	hipFree( dev_c ) ;
   	hipFree( scale ) ;
}